#include "hip/hip_runtime.h"
//
//   v0.2 corrigida por WZola aug/2017 para ficar de acordo com novo wb.h 
//        (ou seja de acordo com wb4.h)
//        

//#include <wb.h>     // original
// DOWNLOAD wb4.h from the discipline site
#include "./wb4.h" // use our new lib, wherever it is
                                              

#include <string.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

//@@ INSERT CODE HERE
__global__ void colorConvert(unsigned char* grayImage,unsigned char* rgbImage,
                             int width, int height, int channels)
{
    int x =  threadIdx.x + blockIdx.x * blockDim.x;
    int y =  threadIdx.y + blockIdx.y * blockDim.y;
    
    if(x < width && y < height)
    {
        int idx = y * width + x;
        unsigned char r =  rgbImage[channels * idx]; // red value for pixel
        unsigned char g =  rgbImage[channels * idx + 1]; // green value for pixel
        unsigned char b =  rgbImage[channels * idx + 2]; // blue value for pixel

        grayImage[idx] = (0.21*r + 0.71*g + 0.07*b);
    }
}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int imageChannels;
  int imageWidth;
  int imageHeight;
  char *inputImageFile;
  wbImage_t inputImage;
  wbImage_t outputImage;

//  float *hostInputImageData;
//  float *hostOutputImageData;
//  float *deviceInputImageData;
//  float *deviceOutputImageData;

  unsigned char *hostInputImageData;
  unsigned char *hostOutputImageData;
  unsigned char *deviceInputImageData;
  unsigned char *deviceOutputImageData;

  args = wbArg_read(argc, argv); /* parse the input arguments */
//  show_args( args ); // debug

//  inputImageFile = wbArg_getInputFileName(args, 2);
    inputImageFile = argv[2];

//  inputImage = wbImportImage(inputImageFile);
  inputImage = wbImport(inputImageFile);

  imageWidth  = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);
  // For this lab the value is always 3
  imageChannels = wbImage_getChannels(inputImage);

  // Since the image is monochromatic, it only contains one channel
  outputImage = wbImage_new(imageWidth, imageHeight, 1);

  hostInputImageData  = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  hipMalloc((void **)&deviceInputImageData,
             imageWidth * imageHeight * imageChannels);
  hipMalloc((void **)&deviceOutputImageData,
             imageWidth * imageHeight);
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  hipMemcpy(deviceInputImageData, hostInputImageData,
             imageWidth * imageHeight * imageChannels,
             hipMemcpyHostToDevice);
  wbTime_stop(Copy, "Copying data to the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ INSERT CODE HERE
  int blockSize = 5;
  dim3 dimGrid((imageWidth-1)/blockSize + 1, (imageHeight-1)/blockSize+1, 1);
  dim3 dimBlock(blockSize, blockSize, 1);
  colorConvert<<<dimGrid,dimBlock>>>(deviceOutputImageData, deviceInputImageData, 
                                   imageWidth, imageHeight, imageChannels);

  
  wbTime_stop(Compute, "Doing the computation on the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Copy, "Copying data from the GPU");
  hipMemcpy(hostOutputImageData, deviceOutputImageData,
             imageWidth * imageHeight,
             hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  wbSolution(args, outputImage);

  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);

  wbImage_delete(outputImage);
  wbImage_delete(inputImage);

  return 0;
}
