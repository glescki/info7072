#include "hip/hip_runtime.h"
// v0.2 modified by WZ

//#include <wb.h>
#include "wb4.h" // use our lib instead (under construction)
#define GTX480   480
#define GTX680   680
#define GTX750Ti 7502
#define GPUmodel GTX750Ti

#if GPUmodel == GTX480
    #define MP 15   // number of mutiprocessors (SMs) in GTX480
    #define GRID1(MP*2) // GRID sizefor rgb2uintKernelSHM and rgb2uintKernelSHM kernels
    #define NT1 768 // number of threads per block in the 
                    //   rgb2uintKernelSHM and rgb2uintKernelSHM kernels
                    //    this is perhaps the best value for GTX480
#elif GPUmodel == GTX680
    #define MP 8    // number of mutiprocessors (SMs) in GTX680
    #define GRID1(MP*2) // GRID sizefor rgb2uintKernelSHM and rgb2uintKernelSHM kernels
    #define NT1 1024    // number of threads per block in the 
                        //   rgb2uintKernelSHM and rgb2uintKernelSHM kernels
                        //    this is perhaps the best value for GTX680
#elif GPUmodel == GTX750ti
    #define MP 5
    #define GRID1(MP*2)
    #define NT1 1024    

#endif

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define BLUR_SIZE 5
#define NTHx 32
#define NTHy 32
#define TILE_WIDTH 64 - 2*BLUR_SIZE
#define TILE_HEIGHT 64 - 2*BLUR_SIZE


//@@ INSERT CODE HERE
__global__ void rgb2uintKernelSHM(  unsigned int* argb, unsigned char* rgb, 
                                   int w, int h )
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if ( x < w && y < h )
  {
      int idx = y * w + x;
      unsigned char r = rgb[idx * 3];
      unsigned char g = rgb[idx * 3 + 1];
      unsigned char b = rgb[idx * 2 + 2];
      unsigned int v = ((unsigned int)r << 16) + ((unsigned int)g << 8) + (unsigned int)b;

      argb[idx] = v;
  }
}

__global__ void uint2rgbKernelSHM(  unsigned int* argb, unsigned char* rgb, 
                                   int w, int h )
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if ( x < w && y < h )
  {
      int idx = y * w + x;
      unsigned char r = (unsigned char) (argb[idx] >> 16 & 0xff);
      unsigned char g = (unsigned char) (argb[idx] >> 8 & 0xff);
      unsigned char b = (unsigned char) (argb[idx] & 0xff);

      rgb[idx * 3] = r;
      rgb[idx * 3 + 1] = g;
      rgb[idx * 3 + 2] = b;
  }
}

__global__ void blurKernelSHM(unsigned char* in, unsigned char* out, int w, int h) 
{
  int bx = blockIdx.x; int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;

  int Col = bx * blockDim.x + tx;
  int Row = by * blockDim.y + ty;

  if (Col > h || Row > w)
      return;
   
  const int tile_size = ( TILE_HEIGHT + 2*BLUR_SIZE ) * ( TILE_WIDTH + 2*BLUR_SIZE );
  __shared__ unsigned char tile[ tile_size ];

  int block_size = NTHx * NTHy;

  for ( int i = 0; i < tile_size / block_size; ++i)
  {
    int tile_idx = (ty * blockDim.y + tx) + i * block_size;
    /* tile[tile_idx] =  */
  }

  /* if ( (Row < w && i * TILE_WIDTH + tx < w) || (Col < h && i * TILE_WIDTH + ty < h) ) */
  /* { */
      /* tile[ty][tx] = in[Row * w + ( i * TILE_WIDTH + tx )]; */
  /* } */
  /* else */
  /* { */
      /* tile[ty][tx] = 0.0; */
  /* } */
  /* __syncthreads(); */

  int pixels = 0;
  for(int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE + 1; ++blurRow)
  {
      for(int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE + 1; ++blurCol)
      {
          int curRow = Row + blurRow;
          int curCol = Col + blurCol;

          if((curRow > -1) && (curRow < h) && (curCol > -1) && (curCol < w))
          {
              int idx = curRow * w + curCol;
              unsigned char r = (unsigned char) (in[idx] >> 16 & 0xff);
              unsigned char g = (unsigned char) (in[idx] >> 8 & 0xff);
              unsigned char b = (unsigned char) (in[idx] & 0xff);

              pixels++;
          }
      }
  }

  /* int idxR = (y * w + x) * 3; */
  /* int idxG = (y * w + x) * 3 + 1; */
  /* int idxB = (y * w + x) * 3 + 2; */
  /* out[idxR] = (unsigned char)(pixValR/pixels); */
  /* out[idxG] = (unsigned char)(pixValG/pixels); */
  /* out[idxB] = (unsigned char)(pixValB/pixels); */

}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int imageWidth;
  int imageHeight;
  char *inputImageFile;
  wbImage_t inputImage;
  wbImage_t outputImage;
  unsigned char *hostInputImageData;
  unsigned char *hostOutputImageData;
  unsigned char *deviceInputImageData;
  unsigned char *deviceOutputImageData;
  unsigned int *deviceInputImageData_argb;
  unsigned int *deviceOutputImageData_argb;

  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 1);
  printf( "imagem de entrada: %s\n", inputImageFile );

  inputImage = wbImport(inputImageFile);

  imageWidth  = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);

// NOW: input and output images are RGB (3 channel)
  outputImage = wbImage_new(imageWidth, imageHeight, 3);

  hostInputImageData  = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  // rgb format image (with pixels as char)
  hipMalloc((void **)&deviceInputImageData,
             imageWidth * imageHeight * sizeof(unsigned char) * 3);
  hipMalloc((void **)&deviceOutputImageData,
             imageWidth * imageHeight * sizeof(unsigned char) * 3);
 
  // argb format image (with pixels as int)
  hipMalloc((void **)&deviceInputImageData,
             imageWidth * imageHeight * sizeof(unsigned int)); 
  hipMalloc((void **)&deviceOutputImageData,
             imageWidth * imageHeight * sizeof(unsigned int));
 
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
 hipMemcpy(deviceInputImageData, hostInputImageData,
            imageWidth * imageHeight * sizeof(unsigned char) * 3,
            hipMemcpyHostToDevice);

  wbTime_stop(Copy, "Copying data to the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Compute, "Doing the computation on the GPU");
  
  dim3 dimGrid((imageWidth-1)/NTHy + 1, (imageHeight-1)/NTHx+1, 1);
  dim3 dimBlock(NTHy, NTHx, 1);


  blurKernelSHM<<<dimGrid,dimBlock>>>(deviceInputImageData, deviceOutputImageData,
                                   imageWidth, imageHeight);

  wbTime_stop(Compute, "Doing the computation on the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Copy, "Copying data from the GPU");
  hipMemcpy(hostOutputImageData, deviceOutputImageData,
             imageWidth * imageHeight * sizeof(unsigned char) * 3,
             hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  wbSolution(args, outputImage);
  // DEBUG: if you want to see your image, 
  //   will generate file bellow in current directory
  wbExport( "blurred.ppm", outputImage );

  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);

  wbImage_delete(outputImage);
  wbImage_delete(inputImage);

  return 0;
}
